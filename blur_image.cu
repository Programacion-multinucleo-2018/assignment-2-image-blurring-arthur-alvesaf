#include "hip/hip_runtime.h"
// Arthur Alves Araujo Ferreira
// A01022593
// Compile with nvcc -o blur blur_image.cu -lopencv_core -lopencv_highgui -lopencv_imgproc

// Includes
#include <iostream>
#include <chrono>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace std;

// input - input image one dimensional array
// ouput - output image one dimensional array
// width, height - width and height of the images
// colorWidthStep - number of color bytes (cols * colors)
// grayWidthStep - number of gray bytes
__global__ void blur_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep)
{
	// 2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	// Only valid threads perform memory I/O
	if ((xIndex < width) && (yIndex < height))
	{
		//Location of colored pixel in input
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);

		// Variable initialization
		char resultantBlue = 0;
		char resultantGreen = 0;
		char resultantRed = 0;
		int shiftIdx = 0;
		// Sum average colors around
		// Iterate horizontally and vertically around color_tid
		for (int xOff = -2; xOff < 3; xOff++) {
			for (int yOff = -2; yOff < 3; yOff++) {
				shiftIdx = color_tid+(xOff*3)+(yOff*width*3);
				resultantBlue += input[shiftIdx]*1/26.f;
				resultantGreen += input[shiftIdx+1]*1/26.f;
				resultantRed += input[shiftIdx+2]*1/26.f;
			}
		}

		// Save resulting pixel to output
		output[color_tid]   = static_cast<unsigned char>(resultantBlue);
		output[color_tid+1] = static_cast<unsigned char>(resultantGreen);
		output[color_tid+2] = static_cast<unsigned char>(resultantRed);
	}
}

// Box blur function given an opencv mat and output
void blur_image(const cv::Mat& input, cv::Mat& output)
{
	cout << "Input image step: " << input.step << " rows: " << input.rows << " cols: " << input.cols << endl;

	// Calculate total number of bytes of input and output image
	// Step = cols * number of colors
	size_t bytes = input.step * input.rows;

	unsigned char *d_input, *d_output;

	// Allocate device memory
	hipMalloc<unsigned char>(&d_input, bytes);
	hipMalloc<unsigned char>(&d_output, bytes);

	// Copy data from OpenCV input image to device memory
	hipMemcpy(d_input, input.ptr(), bytes, hipMemcpyHostToDevice);

	// Specify a reasonable block size
	const dim3 block(16, 16);

	// Calculate grid size to cover the whole image
	// const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);
	const dim3 grid((int)ceil((float)input.cols / block.x), (int)ceil((float)input.rows/ block.y));
	// printf("blur_kernel<<<(%d, %d) , (%d, %d)>>>\n", grid.x, grid.y, block.x, block.y);

	double total = 0;
	//Call the wrapper function
	for (int i = 0; i < 1; i++) {
		auto start = std::chrono::high_resolution_clock::now();
		// blur_image(input, output);
		blur_kernel <<<grid, block >>>(d_input, d_output, input.cols, input.rows, static_cast<int>(input.step), static_cast<int>(output.step));
		auto end = std::chrono::high_resolution_clock::now();
		std::chrono::duration<float, std::milli> duration_ms = end - start;
		total += duration_ms.count();
	}

	cout << "GPU Blur in " << total/1 << " ms." << endl;

	// Synchronize to check for any kernel launch errors
	hipDeviceSynchronize();

	// Copy back data from destination device meory to OpenCV output image
	hipMemcpy(output.ptr(), d_output, bytes, hipMemcpyDeviceToHost);

	// Free the device memory
	hipFree(d_input);
	hipFree(d_output);
}

int main(int argc, char *argv[])
{
	// Variable initialization
	string imagePath;

	// Check for program inputs
	if(argc < 2)
		imagePath = "image.jpg";
  	else
  		imagePath = argv[1];

	// Read input image from the disk
	cv::Mat input = cv::imread(imagePath, CV_LOAD_IMAGE_COLOR);

	if (input.empty())
	{
		cout << "Image Not Found!" << std::endl;
		cin.get();
		return -1;
	}

	//Create output image
	cv::Mat output(input.rows, input.cols, input.type());

	//Call the wrapper function
	blur_image(input, output);

	//Allow the windows to resize
	namedWindow("Input", cv::WINDOW_NORMAL);
	namedWindow("Output", cv::WINDOW_NORMAL);

	//Show the input and output
	imshow("Input", input);
	imshow("Output", output);

	//Wait for key press
	cv::waitKey();

	return 0;
}
